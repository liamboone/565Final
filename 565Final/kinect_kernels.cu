#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cutil_math.h>
#include "cutil_inline.h"

#define DEG_TO_RAD 0.017453292519943 

float4 * dRGBA;
unsigned short * dRAW;
int * dWARP;
int * dBUFF;
int * dHUFF;
int * dEDGE;
int * hmax;

__device__
unsigned short getDepthFromRAW( unsigned short raw )
{
	return ( raw >> 3 );
}

__device__
unsigned short getPlayerFromRAW( unsigned short raw )
{
	return ( raw & 0x7 );
}

__device__
float3 getWorldSpaceFromDepthSpace( int x, int y, short depth, int width, int height, float2 XYScale )
{
	float3 XYZ;
	float phi   = ( (float)x / (float)width  * 57.0f - 28.5f ) * DEG_TO_RAD;
	float theta = ( (float)y / (float)height * 43.0f + 68.5f ) * DEG_TO_RAD;
	XYZ.x = depth;
	XYZ.y = (int)(depth * tan(phi));
	XYZ.z = (int)(depth / tan(theta) / cos(phi));
	return XYZ;
}

__device__
int getIndex( int x, int y, int width, int limx, int limy )
{
	if (x < 0 || y < 0 || x >= limx || y >= limy)
		return -1;
	return width * y + x;
}

__global__
void clear_k( int * warp, int * hough, int * edge, int * maximum, unsigned int width, unsigned int height )
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if( x >= width || y >= height )
	{
		return;
	}

	unsigned int cindex = (height-y-1)*width+x;
	if( cindex == 0 ) *maximum = 0;
	warp[cindex] = 3000;
	hough[cindex] = 0;
	edge[cindex] = 0;
}

__global__
void make_pretty_k( float4 * color, int * warp, int * edge, int* hough, int * maximum, unsigned int width, unsigned int height )
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if( x == width-1 || y == height-1 || x == 0 || y == 0 )
	{
		return;
	}

	unsigned int cindex = (height-y-1)*width+x;
	float fdepth = warp[cindex] == 0 ? 0.0 : ( 1.0 - ( warp[cindex] - 400.0 ) / 2600.0 );
	
	color[cindex] = make_float4(	hough[cindex] >= *maximum, 
									edge[cindex] > 0, 
									0.0,//( fdepth > 0 ? sqrt( (float)min( hough[cindex] / 40.0, 1.0 ) ) : 0.0 ), 
									1.0 );
}

__global__
void max_5x5_k( int * buff, int * warp, unsigned int width, unsigned int height )
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if( x == width-1 || y == height-1 || x == 0 || y == 0 )
	{
		return;
	}

	int index = (height-y-1)*width+x;

	int maximum = warp[index];
	for( int i = -2; i <= 2; i ++ )
	{
		for( int j = -2; j <= 2; j ++ )
		{
			maximum = max( maximum, warp[index+i+j*width] );
		}
	}
	buff[index] = maximum;
}


__global__
void find_head_k( int * maximum, int * warp, int * hough, unsigned int width, unsigned int height )
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if( x == width-1 || y == height-1 || x == 0 || y == 0 )
	{
		return;
	}

	int index = (height-y-1)*width+x;

	if( warp[index] < 3000 ) atomicMax( maximum, hough[index] );
}

__global__
void hough_k( int * buff, int * edge, unsigned int width, unsigned int height )
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if( x == width-1 || y == height-1 || x == 0 || y == 0 )
	{
		return;
	}

	int index = (height-y-1)*width+x;
	
	for( int t = 0; t < 180; t += 10 )
	{
		int xoff = 10.0 * cos( t * DEG_TO_RAD ) + 0.5;
		int yoff = 10.0 * sin( t * DEG_TO_RAD ) + 0.5;

		for( int i = -1; i <= 1; i ++ )
		{
			for( int j = -1; j <= 1; j ++ )
			{
				buff[index] += edge[index+(i+xoff)+(j+yoff)*width];
			}
		}
	}
}

__global__
void min_5x5_k( int * buff, int * warp, unsigned int width, unsigned int height )
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if( x == width-1 || y == height-1 || x == 0 || y == 0 )
	{
		return;
	}

	int index = (height-y-1)*width+x;

	int minimum = warp[index];
	for( int i = -2; i <= 2; i ++ )
	{
		for( int j = -2; j <= 2; j ++ )
		{
			minimum = min( minimum, warp[index+i+j*width] );
		}
	}
	buff[index] = minimum;
}

__global__
void edge_3x3_k( int * buff, int * warp, unsigned int width, unsigned int height )
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if( x == width-1 || y == height-1 || x == 0 || y == 0 )
	{
		return;
	}

	int index = (height-y-1)*width+x;

	int count = 0;
	for( int i = -1; i <= 1; i ++ )
	{
		for( int j = -1; j <= 1; j ++ )
		{
			count += warp[index+i+j*width] < 3000;
		}
	}
	buff[index] = ( count < 7 && warp[index] < 3000 ) ? 1 : 0;
}

__global__
void process_depth_k( int * warp, unsigned short * raw, unsigned int width, unsigned int height, float2 XYScale, int target )
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	int rHeight = 315, rWidth = 434;

	if( x == width-1 || y == height-1 )
	{
		return;
	}
	
	unsigned int index = y*width+x;
	unsigned int cindex = (height-y-1)*width+x;

	unsigned short player = getPlayerFromRAW( raw[index] );
	unsigned short depth = getDepthFromRAW( raw[index] );

	float3 worldpos = getWorldSpaceFromDepthSpace( x, y, depth, width, height, XYScale );

	int wpx = (worldpos.y / 10 ) + rWidth / 2.0;
	int wpy = (worldpos.z / 10) + rHeight / 2.0;
		
	int warpindex = getIndex( wpx+1, wpy+1, width, rWidth+1, rHeight+1 );

	if( warpindex >= 0 && target == player )
		atomicMin( &(warp[warpindex]), (int)depth );
	else
		atomicMin( &(warp[cindex]), (int)3000 );
}

extern "C" void process_depth( dim3 dimGrid, dim3 dimBlock, float4 * depthRGBA, unsigned short * depthRAW, unsigned int width, unsigned int height, float xyscale, int target )
{
	hipMemcpy( dRAW, depthRAW,  width * height * sizeof( unsigned short ), hipMemcpyHostToDevice ); cutilCheckMsg("RAW Depth Transfer");
	clear_k<<< dimGrid, dimBlock >>> ( dWARP, dHUFF, dEDGE, hmax, width, height );
	process_depth_k <<< dimGrid, dimBlock >>> ( dWARP, dRAW, width, height, make_float2( xyscale ), target ); cutilCheckMsg("Depth Process");

	min_5x5_k <<< dimGrid, dimBlock >>> ( dBUFF, dWARP, width, height ); cutilCheckMsg("Min filt");
	max_5x5_k <<< dimGrid, dimBlock >>> ( dWARP, dBUFF, width, height ); cutilCheckMsg("Max filt");
	edge_3x3_k <<< dimGrid, dimBlock >>> ( dEDGE, dWARP, width, height ); cutilCheckMsg("Edge filt");
	hough_k <<< dimGrid, dimBlock >>> ( dHUFF, dEDGE, width, height ); cutilCheckMsg("Hough filt");
	find_head_k <<< dimGrid, dimBlock >>> ( hmax, dWARP, dHUFF, width, height ); cutilCheckMsg("Big Max");
	make_pretty_k <<< dimGrid, dimBlock >>> ( dRGBA, dWARP, dEDGE, dHUFF, hmax, width, height ); cutilCheckMsg("Convert to Image");
	
	hipMemcpy( depthRGBA, dRGBA, width * height * sizeof( float4 ), hipMemcpyDeviceToHost ); cutilCheckMsg("RGBA Depth Transfer");
}

extern "C" void cudaInit( unsigned int width, unsigned int height )
{
	cutilCheckMsg("Before");
	hipMalloc( &dRGBA, width * height * sizeof( float4 ) );
	cutilCheckMsg("CUDA Malloc");
	hipMalloc( &dRAW, width * height * sizeof( unsigned short ) );
	cutilCheckMsg("CUDA Malloc");
	hipMalloc( &dWARP, width * height * sizeof( int ) );
	cutilCheckMsg("CUDA Malloc");
	hipMalloc( &dBUFF, width * height * sizeof( int ) );
	cutilCheckMsg("CUDA Malloc");
	hipMalloc( &dHUFF, width * height * sizeof( int ) );
	cutilCheckMsg("CUDA Malloc");
	hipMalloc( &dEDGE, width * height * sizeof( int ) );
	cutilCheckMsg("CUDA Malloc");
	hipMalloc( &hmax, sizeof( int ) );
	cutilCheckMsg("CUDA Malloc");
}